#include "hip/hip_runtime.h"
#include "PCFG.h"
#include"hip/hip_runtime.h"
#include""
#include <cstring>

using namespace std;

void PriorityQueue::CalProb(PT &pt)
{
    // 计算PriorityQueue里面一个PT的流程如下：
    // 1. 首先需要计算一个PT本身的概率。例如，L6S1的概率为0.15
    // 2. 需要注意的是，Queue里面的PT不是“纯粹的”PT，而是除了最后一个segment以外，全部被value实例化的PT
    // 3. 所以，对于L6S1而言，其在Queue里面的实际PT可能是123456S1，其中“123456”为L6的一个具体value。
    // 4. 这个时候就需要计算123456在L6中出现的概率了。假设123456在所有L6 segment中的概率为0.1，那么123456S1的概率就是0.1*0.15

    // 计算一个PT本身的概率。后续所有具体segment value的概率，直接累乘在这个初始概率值上
    pt.prob = pt.preterm_prob;

    // index: 标注当前segment在PT中的位置
    int index = 0;


    for (int idx : pt.curr_indices)
    {
        // pt.content[index].PrintSeg();
        if (pt.content[index].type == 1)
        {
            // 下面这行代码的意义：
            // pt.content[index]：目前需要计算概率的segment
            // m.FindLetter(seg): 找到一个letter segment在模型中的对应下标
            // m.letters[m.FindLetter(seg)]：一个letter segment在模型中对应的所有统计数据
            // m.letters[m.FindLetter(seg)].ordered_values：一个letter segment在模型中，所有value的总数目
            pt.prob *= m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.letters[m.FindLetter(pt.content[index])].total_freq;
            // cout << m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.letters[m.FindLetter(pt.content[index])].total_freq << endl;
        }
        if (pt.content[index].type == 2)
        {
            pt.prob *= m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.digits[m.FindDigit(pt.content[index])].total_freq;
            // cout << m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.digits[m.FindDigit(pt.content[index])].total_freq << endl;
        }
        if (pt.content[index].type == 3)
        {
            pt.prob *= m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.symbols[m.FindSymbol(pt.content[index])].total_freq;
            // cout << m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.symbols[m.FindSymbol(pt.content[index])].total_freq << endl;
        }
        index += 1;
    }
    // cout << pt.prob << endl;
}

void PriorityQueue::init()
{
    // cout << m.ordered_pts.size() << endl;
    // 用所有可能的PT，按概率降序填满整个优先队列
    for (PT pt : m.ordered_pts)
    {
        for (segment seg : pt.content)
        {
            if (seg.type == 1)
            {
                // 下面这行代码的意义：
                // max_indices用来表示PT中各个segment的可能数目。例如，L6S1中，假设模型统计到了100个L6，那么L6对应的最大下标就是99
                // （但由于后面采用了"<"的比较关系，所以其实max_indices[0]=100）
                // m.FindLetter(seg): 找到一个letter segment在模型中的对应下标
                // m.letters[m.FindLetter(seg)]：一个letter segment在模型中对应的所有统计数据
                // m.letters[m.FindLetter(seg)].ordered_values：一个letter segment在模型中，所有value的总数目
                pt.max_indices.emplace_back(m.letters[m.FindLetter(seg)].ordered_values.size());
            }
            if (seg.type == 2)
            {
                pt.max_indices.emplace_back(m.digits[m.FindDigit(seg)].ordered_values.size());
            }
            if (seg.type == 3)
            {
                pt.max_indices.emplace_back(m.symbols[m.FindSymbol(seg)].ordered_values.size());
            }
        }
        pt.preterm_prob = float(m.preterm_freq[m.FindPT(pt)]) / m.total_preterm;
        // pt.PrintPT();
        // cout << " " << m.preterm_freq[m.FindPT(pt)] << " " << m.total_preterm << " " << pt.preterm_prob << endl;

        // 计算当前pt的概率
        CalProb(pt);
        // 将PT放入优先队列
        priority.emplace_back(pt);
    }
    // cout << "priority size:" << priority.size() << endl;
}

void PriorityQueue::PopNext()
{

    // 对优先队列最前面的PT，首先利用这个PT生成一系列猜测
    Generate(priority.front());

    // 然后需要根据即将出队的PT，生成一系列新的PT
    vector<PT> new_pts = priority.front().NewPTs();
    for (PT pt : new_pts)
    {
        // 计算概率
        CalProb(pt);
        // 接下来的这个循环，作用是根据概率，将新的PT插入到优先队列中
        for (auto iter = priority.begin(); iter != priority.end(); iter++)
        {
            // 对于非队首和队尾的特殊情况
            if (iter != priority.end() - 1 && iter != priority.begin())
            {
                // 判定概率
                if (pt.prob <= iter->prob && pt.prob > (iter + 1)->prob)
                {
                    priority.emplace(iter + 1, pt);
                    break;
                }
            }
            if (iter == priority.end() - 1)
            {
                priority.emplace_back(pt);
                break;
            }
            if (iter == priority.begin() && iter->prob < pt.prob)
            {
                priority.emplace(iter, pt);
                break;
            }
        }
    }

    // 现在队首的PT善后工作已经结束，将其出队（删除）
    priority.erase(priority.begin());
}

// 这个函数你就算看不懂，对并行算法的实现影响也不大
// 当然如果你想做一个基于多优先队列的并行算法，可能得稍微看一看了
vector<PT> PT::NewPTs()
{
    // 存储生成的新PT
    vector<PT> res;

    // 假如这个PT只有一个segment
    // 那么这个segment的所有value在出队前就已经被遍历完毕，并作为猜测输出
    // 因此，所有这个PT可能对应的口令猜测已经遍历完成，无需生成新的PT
    if (content.size() == 1)
    {
        return res;
    }
    else
    {
        // 最初的pivot值。我们将更改位置下标大于等于这个pivot值的segment的值（最后一个segment除外），并且一次只更改一个segment
        // 上面这句话里是不是有没看懂的地方？接着往下看你应该会更明白
        int init_pivot = pivot;

        // 开始遍历所有位置值大于等于init_pivot值的segment
        // 注意i < curr_indices.size() - 1，也就是除去了最后一个segment（这个segment的赋值预留给并行环节）
        for (int i = pivot; i < curr_indices.size() - 1; i += 1)
        {
            // curr_indices: 标记各segment目前的value在模型里对应的下标
            curr_indices[i] += 1;

            // max_indices：标记各segment在模型中一共有多少个value
            if (curr_indices[i] < max_indices[i])
            {
                // 更新pivot值
                pivot = i;
                res.emplace_back(*this);
            }

            // 这个步骤对于你理解pivot的作用、新PT生成的过程而言，至关重要
            curr_indices[i] -= 1;
        }
        pivot = init_pivot;
        return res;
    }

    return res;
}

#define MAX_LEN 32
#define CUDA_THRESHOLD 10000

__global__ void generateGuessesCUDA(const char* prefix, const char* all_values, const int* value_offsets,
                                    int num_values, char* output, int prefix_len, int max_len, int* d_flag)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_values) return;

    const char* val = all_values + value_offsets[idx];
    char* out = output + idx * max_len;

    // Copy prefix
    for (int i = 0; i < prefix_len && i < max_len - 1; ++i)
        out[i] = prefix[i];

    // Copy value
    int j = 0;
    while (val[j] != '\0' && (prefix_len + j) < max_len - 1)
    {
        out[prefix_len + j] = val[j];
        ++j;
    }
    out[prefix_len + j] = '\0';

    if (val[j] != '\0') {
        atomicExch(d_flag, 1);
    }
}

void PriorityQueue::Generate(PT pt)
{
    CalProb(pt);

    string prefix;
    if (pt.content.size() > 1)
    {
        int seg_idx = 0;
        for (int idx : pt.curr_indices) 
        {
            if (seg_idx >= pt.content.size() - 1) break;

            if (pt.content[seg_idx].type == 1)
                prefix += m.letters[m.FindLetter(pt.content[seg_idx])].ordered_values[idx];

            if (pt.content[seg_idx].type == 2)
                prefix += m.digits[m.FindDigit(pt.content[seg_idx])].ordered_values[idx];

            if (pt.content[seg_idx].type == 3)
                prefix += m.symbols[m.FindSymbol(pt.content[seg_idx])].ordered_values[idx];

            seg_idx++;
        }
    }

    segment* a;
    if (pt.content.back().type == 1)
        a = &m.letters[m.FindLetter(pt.content.back())];
    if (pt.content.back().type == 2)
        a = &m.digits[m.FindDigit(pt.content.back())];
    if (pt.content.back().type == 3)
        a = &m.symbols[m.FindSymbol(pt.content.back())];

    int num = pt.max_indices.back();

    // 如果猜测数量较小，直接用CPU串行生成所有猜测，避免GPU启动开销
    if (num < CUDA_THRESHOLD) 
    {
        // cout<<"[INFO] Use CPU"<<endl;
        for (int i = 0; i < num; ++i)
        {
            string g = prefix + a->ordered_values[i];
            guesses.emplace_back(g);
            total_guesses += 1;
        }
        return;
    }

    // cout<<"[INFO] Use GPU"<<endl;

    // 1. 分配并拷贝prefix字符串到GPU内存
    char* d_prefix;
    hipMalloc(&d_prefix, prefix.size());
    hipMemcpy(d_prefix, prefix.c_str(), prefix.size(), hipMemcpyHostToDevice);

    // 2. 计算所有最后一个segment字符串的总长度（含结束符）
    size_t total_value_len = 0;
    for (int i = 0; i < num; ++i)
        total_value_len += a->ordered_values[i].size() + 1;

    // 3. 在CPU端连续分配内存，拷贝所有字符串并记录偏移
    int* value_offsets = new int[num];
    char* temp_all_values = new char[total_value_len];
    int offset = 0;
    for (int i = 0; i < num; ++i)
    {
        strcpy(temp_all_values + offset, a->ordered_values[i].c_str());
        value_offsets[i] = offset;
        offset += a->ordered_values[i].size() + 1;
    }

    // 4. 分配GPU内存并拷贝字符串数据
    char* d_all_values;
    hipMalloc(&d_all_values, total_value_len);
    hipMemcpy(d_all_values, temp_all_values, total_value_len, hipMemcpyHostToDevice);

    // 5. 分配GPU内存并拷贝偏移数组
    int* d_offsets;
    hipMalloc(&d_offsets, sizeof(int) * num);
    hipMemcpy(d_offsets, value_offsets, sizeof(int) * num, hipMemcpyHostToDevice);

    // 6. 分配GPU输出缓冲区，用于存储每条猜测字符串，长度为num * MAX_LEN
    char* d_output;
    hipMalloc(&d_output, num * MAX_LEN);

    // 7. 分配GPU设备标志位内存（用于检测超长字符串等异常，初始化为0）
    int* d_flag;
    int h_flag = 0;
    hipMalloc(&d_flag, sizeof(int));
    hipMemcpy(d_flag, &h_flag, sizeof(int), hipMemcpyHostToDevice);

    // 8. 启动CUDA核函数，执行猜测生成，线程块大小256，计算块数确保覆盖所有猜测
    int threads = 256;
    int blocks = (num + threads - 1) / threads;
    generateGuessesCUDA<<<blocks, threads>>>(d_prefix, d_all_values, d_offsets, num, d_output,
                                             prefix.size(), MAX_LEN, d_flag);
    hipDeviceSynchronize();
    
    // 9. 拷贝设备标志位回CPU，检测是否有猜测超长被截断
    hipMemcpy(&h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
    if (h_flag) 
    {
        cerr << "[Warning] One or more guesses exceeded MAX_LEN and were truncated." << endl;
    }

    // 10. 从GPU拷贝生成的所有猜测回CPU内存
    char* h_output = new char[num * MAX_LEN];
    hipMemcpy(h_output, d_output, num * MAX_LEN, hipMemcpyDeviceToHost);

    // 11. 逐条将GPU生成的猜测存入全局猜测结果容器
    for (int i = 0; i < num; ++i)
    {
        guesses.emplace_back(h_output + i * MAX_LEN);
        total_guesses += 1;
    }

    // 12. 释放所有申请的GPU和CPU内存，防止内存泄漏
    hipFree(d_prefix);
    hipFree(d_all_values);
    hipFree(d_offsets);
    hipFree(d_output);
    hipFree(d_flag);
    delete[] temp_all_values;
    delete[] value_offsets;
    delete[] h_output;
}